#include "hip/hip_runtime.h"
/******************************************************************************
*       SOFA, Simulation Open-Framework Architecture, version 1.0 beta 3      *
*                (c) 2006-2008 MGH, INRIA, USTL, UJF, CNRS                    *
*                                                                             *
* This library is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This library is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this library; if not, write to the Free Software Foundation,     *
* Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301 USA.          *
*******************************************************************************
*                               SOFA :: Modules                               *
*                                                                             *
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include "CudaCommon.h"
#include "CudaMath.h"
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION != 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    void UniformMassCuda3f_addMDx(unsigned int size, float mass, void* res, const void* dx);
    void UniformMassCuda3f_accFromF(unsigned int size, float mass, void* a, const void* f);
    void UniformMassCuda3f_addForce(unsigned int size, const float *mg, void* f);

    void UniformMassCuda3f1_addMDx(unsigned int size, float mass, void* res, const void* dx);
    void UniformMassCuda3f1_accFromF(unsigned int size, float mass, void* a, const void* f);
    void UniformMassCuda3f1_addForce(unsigned int size, const float *mg, void* f);

#ifdef SOFA_DEV
#ifdef SOFA_GPU_CUDA_DOUBLE

    void UniformMassCuda3d_addMDx(unsigned int size, double mass, void* res, const void* dx);
    void UniformMassCuda3d_accFromF(unsigned int size, double mass, void* a, const void* f);
    void UniformMassCuda3d_addForce(unsigned int size, const double *mg, void* f);

    void UniformMassCuda3d1_addMDx(unsigned int size, double mass, void* res, const void* dx);
    void UniformMassCuda3d1_accFromF(unsigned int size, double mass, void* a, const void* f);
    void UniformMassCuda3d1_addForce(unsigned int size, const double *mg, void* f);

#endif // SOFA_GPU_CUDA_DOUBLE
#endif // SOFA_DEV

}

//////////////////////
// GPU-side methods //
//////////////////////

template<class real>
__global__ void UniformMassCuda1t_addMDx_kernel(int size, const real mass, real* res, const real* dx)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size)
    {
        res[index] += dx[index] * mass;
    }
}

template<class real>
__global__ void UniformMassCuda3t_addMDx_kernel(int size, const real mass, CudaVec3<real>* res, const CudaVec3<real>* dx)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size)
    {
        //res[index] += dx[index] * mass;
        CudaVec3<real> dxi = dx[index];
        CudaVec3<real> ri = res[index];
        ri += dxi * mass;
        res[index] = ri;
    }
}

template<class real>
__global__ void UniformMassCuda3t1_addMDx_kernel(int size, const real mass, CudaVec4<real>* res, const CudaVec4<real>* dx)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size)
    {
        //res[index] += dx[index] * mass;
        CudaVec4<real> dxi = dx[index];
        CudaVec4<real> ri = res[index];
        ri.x += dxi.x * mass;
        ri.y += dxi.y * mass;
        ri.z += dxi.z * mass;
        res[index] = ri;
    }
}

template<class real>
__global__ void UniformMassCuda1t_accFromF_kernel(int size, const real inv_mass, real* a, const real* f)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size)
    {
        a[index] = f[index] * inv_mass;
    }
}

template<class real>
__global__ void UniformMassCuda3t_accFromF_kernel(int size, const real inv_mass, CudaVec3<real>* a, const CudaVec3<real>* f)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size)
    {
        //a[index] = f[index] * inv_mass;
        CudaVec3<real> fi = f[index];
        fi *= inv_mass;
        a[index] = fi;
    }
}

template<class real>
__global__ void UniformMassCuda3t1_accFromF_kernel(int size, const real inv_mass, CudaVec4<real>* a, const CudaVec4<real>* f)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size)
    {
        //a[index] = f[index] * inv_mass;
        CudaVec4<real> fi = f[index];
        fi.x *= inv_mass;
        fi.y *= inv_mass;
        fi.z *= inv_mass;
        a[index] = fi;
    }
}

template<class real>
__global__ void UniformMassCuda1t_addForce_kernel(int size, const real mg, real* f)
{
    int index = umul24(blockIdx.x,BSIZE);
    if (index < size)
    {
        f[index] += mg;
    }
}

template<class real>
__global__ void UniformMassCuda3t_addForce_kernel(int size, const CudaVec3<real> mg, real* f)
{
    //int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //f[index] += mg;
    f += umul24(blockIdx.x,BSIZE*3); //blockIdx.x*BSIZE*3;
    int index = threadIdx.x;
    //! Dynamically allocated shared memory to reorder global memory access
    extern  __shared__  real temp[];
    temp[index] = f[index];
    temp[index+BSIZE] = f[index+BSIZE];
    temp[index+2*BSIZE] = f[index+2*BSIZE];

    __syncthreads();

    if (umul24(blockIdx.x,BSIZE)+threadIdx.x < size)
    {
        int index3 = umul24(index,3); //3*index;
        temp[index3+0] += mg.x;
        temp[index3+1] += mg.y;
        temp[index3+2] += mg.z;
    }

    __syncthreads();

    f[index] = temp[index];
    f[index+BSIZE] = temp[index+BSIZE];
    f[index+2*BSIZE] = temp[index+2*BSIZE];
}

template<class real>
__global__ void UniformMassCuda3t1_addForce_kernel(int size, const CudaVec3<real> mg, CudaVec4<real>* f)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size)
    {
        //f[index] += mg;
        CudaVec4<real> fi = f[index];
        fi.x += mg.x;
        fi.y += mg.y;
        fi.z += mg.z;
        f[index] = fi;
    }
}

//////////////////////
// CPU-side methods //
//////////////////////

void UniformMassCuda3f_addMDx(unsigned int size, float mass, void* res, const void* dx)
{
    dim3 threads(BSIZE,1);
    //dim3 grid((size+BSIZE-1)/BSIZE,1);
    //UniformMassCuda3t_addMDx_kernel<float><<< grid, threads >>>(size, mass, (CudaVec3<float>*)res, (const CudaVec3<float>*)dx);
    dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    UniformMassCuda1t_addMDx_kernel<float><<< grid, threads >>>(3*size, mass, (float*)res, (const float*)dx);
}

void UniformMassCuda3f1_addMDx(unsigned int size, float mass, void* res, const void* dx)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    UniformMassCuda3t1_addMDx_kernel<float><<< grid, threads >>>(size, mass, (CudaVec4<float>*)res, (const CudaVec4<float>*)dx);
    //dim3 grid((4*size+BSIZE-1)/BSIZE,1);
    //UniformMassCuda1t_addMDx_kernel<float><<< grid, threads >>>(4*size, mass, (float*)res, (const float*)dx);
}

void UniformMassCuda3f_accFromF(unsigned int size, float mass, void* a, const void* f)
{
    dim3 threads(BSIZE,1);
    //dim3 grid((size+BSIZE-1)/BSIZE,1);
    //UniformMassCuda3t_accFromF_kernel<float><<< grid, threads >>>(size, 1.0f/mass, (CudaVec3<float>*)a, (const CudaVec3<float>*)f);
    dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    UniformMassCuda1t_accFromF_kernel<float><<< grid, threads >>>(3*size, 1.0f/mass, (float*)a, (const float*)f);
}

void UniformMassCuda3f1_accFromF(unsigned int size, float mass, void* a, const void* f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    UniformMassCuda3t1_accFromF_kernel<float><<< grid, threads >>>(size, 1.0f/mass, (CudaVec4<float>*)a, (const CudaVec4<float>*)f);
    //dim3 grid((4*size+BSIZE-1)/BSIZE,1);
    //UniformMassCuda1t_accFromF_kernel<float><<< grid, threads >>>(4*size, 1.0f/mass, (float*)a, (const float*)f);
}

void UniformMassCuda3f_addForce(unsigned int size, const float *mg, void* f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    UniformMassCuda3t_addForce_kernel<float><<< grid, threads, BSIZE*3*sizeof(float) >>>(size, CudaVec3<float>::make(mg[0],mg[1],mg[2]), (float*)f);
}

void UniformMassCuda3f1_addForce(unsigned int size, const float *mg, void* f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    UniformMassCuda3t1_addForce_kernel<float><<< grid, threads >>>(size, CudaVec3<float>::make(mg[0],mg[1],mg[2]), (CudaVec4<float>*)f);
}

#ifdef SOFA_DEV
#ifdef SOFA_GPU_CUDA_DOUBLE

void UniformMassCuda3d_addMDx(unsigned int size, double mass, void* res, const void* dx)
{
    dim3 threads(BSIZE,1);
    //dim3 grid((size+BSIZE-1)/BSIZE,1);
    //UniformMassCuda3t_addMDx_kernel<double><<< grid, threads >>>(size, mass, (CudaVec3<double>*)res, (const CudaVec3<double>*)dx);
    dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    UniformMassCuda1t_addMDx_kernel<double><<< grid, threads >>>(3*size, mass, (double*)res, (const double*)dx);
}

void UniformMassCuda3d1_addMDx(unsigned int size, double mass, void* res, const void* dx)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    UniformMassCuda3t1_addMDx_kernel<double><<< grid, threads >>>(size, mass, (CudaVec4<double>*)res, (const CudaVec4<double>*)dx);
    //dim3 grid((4*size+BSIZE-1)/BSIZE,1);
    //UniformMassCuda1t_addMDx_kernel<double><<< grid, threads >>>(4*size, mass, (double*)res, (const double*)dx);
}

void UniformMassCuda3d_accFromF(unsigned int size, double mass, void* a, const void* f)
{
    dim3 threads(BSIZE,1);
    //dim3 grid((size+BSIZE-1)/BSIZE,1);
    //UniformMassCuda3t_accFromF_kernel<double><<< grid, threads >>>(size, 1.0f/mass, (CudaVec3<double>*)a, (const CudaVec3<double>*)f);
    dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    UniformMassCuda1t_accFromF_kernel<double><<< grid, threads >>>(3*size, 1.0f/mass, (double*)a, (const double*)f);
}

void UniformMassCuda3d1_accFromF(unsigned int size, double mass, void* a, const void* f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    UniformMassCuda3t1_accFromF_kernel<double><<< grid, threads >>>(size, 1.0f/mass, (CudaVec4<double>*)a, (const CudaVec4<double>*)f);
    //dim3 grid((4*size+BSIZE-1)/BSIZE,1);
    //UniformMassCuda1t_accFromF_kernel<double><<< grid, threads >>>(4*size, 1.0f/mass, (double*)a, (const double*)f);
}

void UniformMassCuda3d_addForce(unsigned int size, const double *mg, void* f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    UniformMassCuda3t_addForce_kernel<double><<< grid, threads, BSIZE*3*sizeof(double) >>>(size, CudaVec3<double>::make(mg[0],mg[1],mg[2]), (double*)f);
}

void UniformMassCuda3d1_addForce(unsigned int size, const double *mg, void* f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    UniformMassCuda3t1_addForce_kernel<double><<< grid, threads >>>(size, CudaVec3<double>::make(mg[0],mg[1],mg[2]), (CudaVec4<double>*)f);
}

#endif // SOFA_GPU_CUDA_DOUBLE
#endif // SOFA_DEV

#if defined(__cplusplus) && CUDA_VERSION != 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
