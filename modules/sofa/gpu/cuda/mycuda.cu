/******************************************************************************
*       SOFA, Simulation Open-Framework Architecture, version 1.0 beta 4      *
*                (c) 2006-2009 MGH, INRIA, USTL, UJF, CNRS                    *
*                                                                             *
* This library is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This library is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this library; if not, write to the Free Software Foundation,     *
* Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301 USA.          *
*******************************************************************************
*                               SOFA :: Modules                               *
*                                                                             *
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include "mycuda.h"
#include <hip/hip_runtime.h>
#ifdef WIN32
#include <sofa/helper/system/gl.h>
#endif
#include <cuda_gl_interop.h>

#include <sofa/helper/BackTrace.h>

//#define NO_CUDA

#if defined(__cplusplus)
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

#ifdef NO_CUDA

bool cudaCheck(hipError_t, const char*)
{
    return true;
}

bool cudaInitCalled = false;

int mycudaInit(int)
{
    cudaInitCalled = true;
    return 0;
}

void mycudaMalloc(void **devPtr, size_t)
{
    *devPtr = NULL;
}

void mycudaMallocPitch(void **devPtr, size_t*, size_t, size_t)
{
    *devPtr = NULL;
}

void mycudaFree(void *)
{
}

void mycudaMallocHost(void **hostPtr, size_t size)
{
    *hostPtr = malloc(size);
}

void mycudaFreeHost(void *hostPtr)
{
    free(hostPtr);
}

void mycudaMemcpyHostToDevice(void *, const void *, size_t)
{
}

void mycudaMemcpyDeviceToDevice(void *, const void *, size_t)
{
}

void mycudaMemcpyDeviceToHost(void *, const void *, size_t)
{
}

void mycudaMemcpyHostToDevice2D(void *, size_t, const void *, size_t, size_t, size_t)
{
}

void mycudaMemcpyDeviceToDevice2D(void *, size_t, const void *, size_t, size_t, size_t )
{
}

void mycudaMemcpyDeviceToHost2D(void *, size_t, const void *, size_t, size_t, size_t)
{
}

void mycudaGLRegisterBufferObject(int)
{
}

void mycudaGLUnregisterBufferObject(int)
{
}

void mycudaGLMapBufferObject(void** ptr, int)
{
    *ptr = NULL;
}

void mycudaGLUnmapBufferObject(int)
{
}

#else

bool cudaCheck(hipError_t err, const char* src="?")
{
    if (err == hipSuccess) return true;
    //fprintf(stderr, "CUDA: Error %d returned from %s.\n",(int)err,src);
    mycudaLogError(hipGetErrorString(err), src);
    sofa::helper::BackTrace::dump();
    return false;
}

bool cudaInitCalled = false;

int mycudaInit(int device)
{
    int deviceCount = 0;
    cudaInitCalled = true;
    {
        const char* var = mygetenv("CUDA_MULTIOPS");
        if (var && *var)
        {
            mycudaMultiOpMax = atoi(var);
            if (mycudaMultiOpMax)
                myprintf("CUDA: Merging of up to %d identical operations enabled.\n", mycudaMultiOpMax);
            else
                myprintf("CUDA: Merging of identical operations disabled.\n", mycudaMultiOpMax);
        }
    }
    cudaCheck(hipGetDeviceCount(&deviceCount),"hipGetDeviceCount");
    myprintf("CUDA: %d device(s) found.\n", deviceCount);
    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t dev
#ifdef cudaDevicePropDontCare
            = cudaDevicePropDontCare
#endif
                    ;
        //memset(&dev,0,sizeof(dev));
        //dev.name=NULL;
        //dev.bytes=0;
        //dev.major=0;
        //dev.minor=0;
        cudaCheck(hipGetDeviceProperties(&dev,i),"hipGetDeviceProperties");
#if CUDA_VERSION >= 2010
        myprintf("CUDA:  %d : \"%s\", %d MB, %d cores at %.3f GHz, revision %d.%d",i,dev.name, dev.totalGlobalMem/(1024*1024), dev.multiProcessorCount*8, dev.clockRate * 1e-6f, dev.major, dev.minor);
        if (dev.kernelExecTimeoutEnabled)
            myprintf(", timeout enabled", dev.kernelExecTimeoutEnabled);
        myprintf("\n");
#elif CUDA_VERSION >= 2000
        myprintf("CUDA:  %d : \"%s\", %d MB, %d cores at %.3f GHz, revision %d.%d\n",i,dev.name, dev.totalGlobalMem/(1024*1024), dev.multiProcessorCount*8, dev.clockRate * 1e-6f, dev.major, dev.minor);
#else //if CUDA_VERSION >= 1000
        myprintf("CUDA:  %d : \"%s\", %d MB, cores at %.3f GHz, revision %d.%d\n",i,dev.name, dev.totalGlobalMem/(1024*1024), dev.clockRate * 1e-6f, dev.major, dev.minor);
//#else
//		myprintf("CUDA:  %d : \"%s\", %d MB, revision %d.%d\n",i,(dev.name==NULL?"":dev.name), dev.bytes/(1024*1024), dev.major, dev.minor);
#endif
    }
    if (device==-1)
    {
        const char* var = mygetenv("CUDA_DEVICE");
        device = (var && *var) ? atoi(var):0;
    }
    if (device >= deviceCount)
    {
        myprintf("CUDA: Device %d not found.\n", device);
        return 0;
    }
    else
    {
        hipDeviceProp_t dev;
        cudaCheck(hipGetDeviceProperties(&dev,device));
        myprintf("CUDA: Using device %d : \"%s\"\n",device,dev.name);
        cudaCheck(hipSetDevice(device));
        return 1;
    }

}

void mycudaMalloc(void **devPtr, size_t size)
{
    if (!cudaInitCalled) mycudaInit();
    if (mycudaVerboseLevel>=LOG_INFO) myprintf("CUDA: malloc(%d).\n",size);
    cudaCheck(hipMalloc(devPtr, size),"hipMalloc");
    if (mycudaVerboseLevel>=LOG_TRACE) myprintf("CUDA: malloc(%d) -> 0x%x.\n",size, *devPtr);
}

void mycudaMallocPitch(void **devPtr, size_t* pitch, size_t width, size_t height)
{
    if (!cudaInitCalled) mycudaInit();
    if (mycudaVerboseLevel>=LOG_INFO) myprintf("CUDA: mallocPitch(%d,%d).\n",width,height);
    cudaCheck(hipMallocPitch(devPtr, pitch, width, height),"cudaMalloc2D");
    if (mycudaVerboseLevel>=LOG_TRACE) myprintf("CUDA: mallocPitch(%d,%d) -> 0x%x at pitch %d.\n",width,height, *devPtr, (int)*pitch);
}

void mycudaFree(void *devPtr)
{
    if (mycudaVerboseLevel>=LOG_TRACE) myprintf("CUDA: free(0x%x).\n",devPtr);
    cudaCheck(hipFree(devPtr),"hipFree");
}

void mycudaMallocHost(void **hostPtr, size_t size)
{
    if (!cudaInitCalled) mycudaInit();
    if (mycudaVerboseLevel>=LOG_TRACE) myprintf("CUDA: mallocHost(%d).\n",size);
    cudaCheck(hipHostMalloc(hostPtr, size),"hipHostMalloc");
    if (mycudaVerboseLevel>=LOG_TRACE) myprintf("CUDA: mallocHost(%d) -> 0x%x.\n",size, *hostPtr);
}

void mycudaFreeHost(void *hostPtr)
{
    if (mycudaVerboseLevel>=LOG_TRACE) myprintf("CUDA: freeHost(0x%x).\n",hostPtr);
    cudaCheck(hipHostFree(hostPtr),"hipHostFree");
}

void mycudaMemcpyHostToDevice(void *dst, const void *src, size_t count)
{
    if (!cudaCheck(hipMemcpy(dst, src, count, hipMemcpyHostToDevice),"hipMemcpyHostToDevice"))
        myprintf("in mycudaMemcpyHostToDevice(0x%x, 0x%x, %d)\n",dst,src,count);
}

void mycudaMemcpyDeviceToDevice(void *dst, const void *src, size_t count)
{
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice),"hipMemcpyDeviceToDevice");
}

void mycudaMemcpyDeviceToHost(void *dst, const void *src, size_t count)
{
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost),"hipMemcpyDeviceToHost");
}

void mycudaMemcpyHostToDevice2D(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height)
{
    cudaCheck(hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyHostToDevice),"cudaMemcpyHostToDevice2D");
}

void mycudaMemcpyDeviceToDevice2D(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height)
{
    cudaCheck(hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyDeviceToDevice),"cudaMemcpyDeviceToDevice2D");
}

void mycudaMemcpyDeviceToHost2D(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height)
{
    cudaCheck(hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyDeviceToHost),"cudaMemcpyDeviceToHost2D");
}

void mycudaGLRegisterBufferObject(int id)
{
    if (!cudaInitCalled) mycudaInit();
    cudaCheck(cudaGLRegisterBufferObject((GLuint)id),"cudaGLRegisterBufferObject");
}

void mycudaGLUnregisterBufferObject(int id)
{
    cudaCheck(cudaGLUnregisterBufferObject((GLuint)id),"cudaGLUnregisterBufferObject");
}

void mycudaGLMapBufferObject(void** ptr, int id)
{
    cudaCheck(cudaGLMapBufferObject(ptr, (GLuint)id),"cudaGLMapBufferObject");
}

void mycudaGLUnmapBufferObject(int id)
{
    cudaCheck(cudaGLUnmapBufferObject((GLuint)id),"cudaGLUnmapBufferObject");
}

#endif

#if defined(__cplusplus)
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
