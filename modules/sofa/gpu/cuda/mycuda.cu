#include "mycuda.h"
#include <cuda_gl_interop.h>
#if defined(__cplusplus)
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

void cudaCheck(hipError_t err, const char* src="?")
{
    if (err == hipSuccess) return;
    //fprintf(stderr, "CUDA: Error %d returned from %s.\n",(int)err,src);
    mycudaLogError(err, src);
}

bool cudaInitCalled = false;

int mycudaInit(int device)
{
    int deviceCount = 0;
    cudaInitCalled = true;
    cudaCheck(hipGetDeviceCount(&deviceCount));
    myprintf("CUDA: %d devices found.\n", deviceCount);
    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t dev;
        memset(&dev,0,sizeof(dev));
        //dev.name=NULL;
        //dev.bytes=0;
        //dev.major=0;
        //dev.minor=0;
        cudaCheck(hipGetDeviceProperties(&dev,i));
        //myprintf("CUDA:  %d : \"%s\", %d MB, revision %d.%d\n",i,(dev.name==NULL?"":dev.name), dev.bytes/(1024*1024), dev.major, dev.minor);
        myprintf("CUDA:  %d : \"%s\", %d MB, revision %d.%d\n",i,dev.name, dev.totalGlobalMem/(1024*1024), dev.major, dev.minor);
    }
    if (device >= deviceCount)
    {
        myprintf("CUDA: Device %d not found.\n", device);
        return 0;
    }
    else
    {
        cudaCheck(hipSetDevice(device));
        return 1;
    }
}

void mycudaMalloc(void **devPtr, size_t size)
{
    if (!cudaInitCalled) mycudaInit(0);
    myprintf("CUDA: malloc(%d).\n",size);
    cudaCheck(hipMalloc(devPtr, size),"hipMalloc");
}

void mycudaFree(void *devPtr)
{
    myprintf("CUDA: free().\n");
    cudaCheck(hipFree(devPtr),"hipFree");
}

void mycudaMallocHost(void **hostPtr, size_t size)
{
    if (!cudaInitCalled) mycudaInit(0);
    myprintf("CUDA: mallocHost(%d).\n",size);
    cudaCheck(hipHostMalloc(hostPtr, size),"hipHostMalloc");
}

void mycudaFreeHost(void *hostPtr)
{
    myprintf("CUDA: freeHost().\n");
    cudaCheck(hipHostFree(hostPtr),"hipHostFree");
}

void mycudaMemcpyHostToDevice(void *dst, const void *src, size_t count)
{
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyHostToDevice),"hipMemcpyHostToDevice");
}

void mycudaMemcpyDeviceToDevice(void *dst, const void *src, size_t count)
{
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice),"hipMemcpyDeviceToDevice");
}

void mycudaMemcpyDeviceToHost(void *dst, const void *src, size_t count)
{
    cudaCheck(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost),"hipMemcpyDeviceToHost");
}

void mycudaGLRegisterBufferObject(int id)
{
    if (!cudaInitCalled) mycudaInit(0);
    cudaCheck(cudaGLRegisterBufferObject((GLuint)id),"cudaGLRegisterBufferObject");
}

void mycudaGLUnregisterBufferObject(int id)
{
    cudaCheck(cudaGLUnregisterBufferObject((GLuint)id),"cudaGLUnregisterBufferObject");
}

void mycudaGLMapBufferObject(void** ptr, int id)
{
    cudaCheck(cudaGLMapBufferObject(ptr, (GLuint)id),"cudaGLMapBufferObject");
}

void mycudaGLUnmapBufferObject(int id)
{
    cudaCheck(cudaGLUnmapBufferObject((GLuint)id),"cudaGLUnmapBufferObject");
}

#if defined(__cplusplus)
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
