#include "hip/hip_runtime.h"
/******************************************************************************
*       SOFA, Simulation Open-Framework Architecture, version 1.0 beta 3      *
*                (c) 2006-2008 MGH, INRIA, USTL, UJF, CNRS                    *
*                                                                             *
* This library is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This library is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this library; if not, write to the Free Software Foundation,     *
* Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301 USA.          *
*******************************************************************************
*                               SOFA :: Modules                               *
*                                                                             *
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include "CudaCommon.h"
#include "CudaMath.h"
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    void RegularGridMapperCuda3f_apply(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in);
    void RegularGridMapperCuda3f_applyJ(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in);
    void RegularGridMapperCuda3f_applyJT(unsigned int insize, unsigned int maxNOut, const unsigned int* gridsize, const void* mapT, void* out, const void* in);

    void RegularGridMapperCuda3f1_apply(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in);
    void RegularGridMapperCuda3f1_applyJ(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in);
    void RegularGridMapperCuda3f1_applyJT(unsigned int insize, unsigned int maxNOut, const unsigned int* gridsize, const void* mapT, void* out, const void* in);

    void RegularGridMapperCuda3f_3f1_apply(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in);
    void RegularGridMapperCuda3f_3f1_applyJ(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in);
    void RegularGridMapperCuda3f_3f1_applyJT(unsigned int insize, unsigned int maxNOut, const unsigned int* gridsize, const void* mapT, void* out, const void* in);

    void RegularGridMapperCuda3f1_3f_apply(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in);
    void RegularGridMapperCuda3f1_3f_applyJ(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in);
    void RegularGridMapperCuda3f1_3f_applyJT(unsigned int insize, unsigned int maxNOut, const unsigned int* gridsize, const void* mapT, void* out, const void* in);


    void MeshMapperCuda3f_apply(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in);
    void MeshMapperCuda3f1_apply(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in);
    void MeshMapperCuda3f_3f1_apply(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in);
    void MeshMapperCuda3f1_3f_apply(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in);

    void MeshMapperCuda3f_applyPEq(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in);
    void MeshMapperCuda3f1_applyPEq(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in);
    void MeshMapperCuda3f_3f1_applyPEq(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in);
    void MeshMapperCuda3f1_3f_applyPEq(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in);
}

struct __align__(16) GPUCubeData
{
    int i;
    float fx,fy,fz;
};

struct __align__(8) GPULinearMap
{
    int i;
    float f;
};

//////////////////////
// GPU-side methods //
//////////////////////

template<class TIn>
__global__ void RegularGridMapperCuda3f_apply_kernel(unsigned int size, unsigned int nx, unsigned int nxny, const GPUCubeData* map, float* out, const TIn* in)
{
    const int index0 = umul24(blockIdx.x,BSIZE); //blockDim.x;
    const int index1 = threadIdx.x;

    //! Dynamically allocated shared memory to reorder global memory access
    extern  __shared__  float temp[];

    CudaVec3<float> res = CudaVec3<float>::make(0,0,0);

    GPUCubeData c = map[index0+index1];
    if (index0+index1 < size)
    {
        //const Real fx = map[i].baryCoords[0];
        //const Real fy = map[i].baryCoords[1];
        //const Real fz = map[i].baryCoords[2];
        //out[i] = in[cube[0]] * ((1-fx) * (1-fy) * (1-fz))
        //       + in[cube[1]] * ((  fx) * (1-fy) * (1-fz))
        //       + in[cube[2]] * ((1-fx) * (  fy) * (1-fz))
        //       + in[cube[3]] * ((  fx) * (  fy) * (1-fz))
        //       + in[cube[4]] * ((1-fx) * (1-fy) * (  fz))
        //       + in[cube[5]] * ((  fx) * (1-fy) * (  fz))
        //       + in[cube[6]] * ((1-fx) * (  fy) * (  fz))
        //       + in[cube[7]] * ((  fx) * (  fy) * (  fz));

        res = CudaVec3<float>::make(in [c.i          ]) * ((1-c.fx) * (1-c.fy) * (1-c.fz))
                + CudaVec3<float>::make(in [c.i+1        ]) * ((  c.fx) * (1-c.fy) * (1-c.fz))
                + CudaVec3<float>::make(in [c.i  +nx     ]) * ((1-c.fx) * (  c.fy) * (1-c.fz))
                + CudaVec3<float>::make(in [c.i+1+nx     ]) * ((  c.fx) * (  c.fy) * (1-c.fz))
                + CudaVec3<float>::make(in [c.i     +nxny]) * ((1-c.fx) * (1-c.fy) * (  c.fz))
                + CudaVec3<float>::make(in [c.i+1   +nxny]) * ((  c.fx) * (1-c.fy) * (  c.fz))
                + CudaVec3<float>::make(in [c.i  +nx+nxny]) * ((1-c.fx) * (  c.fy) * (  c.fz))
                + CudaVec3<float>::make(in [c.i+1+nx+nxny]) * ((  c.fx) * (  c.fy) * (  c.fz));
    }

    //__syncthreads();

    const int index3 = umul24(3,index1);

    temp[index3  ] = res.x;
    temp[index3+1] = res.y;
    temp[index3+2] = res.z;

    __syncthreads();

    out += umul24(index0,3);
    out[index1        ] = temp[index1        ];
    out[index1+  BSIZE] = temp[index1+  BSIZE];
    out[index1+2*BSIZE] = temp[index1+2*BSIZE];
}

template<class TIn>
__global__ void RegularGridMapperCuda3f1_apply_kernel(unsigned int size, unsigned int nx, unsigned int nxny, const GPUCubeData* map, CudaVec4<float>* out, const TIn* in)
{
    const int index = umul24(blockIdx.x,BSIZE) + threadIdx.x;

    CudaVec3<float> res = CudaVec3<float>::make(0,0,0);

    GPUCubeData c = map[index];
    if (index < size)
    {
        //const Real fx = map[i].baryCoords[0];
        //const Real fy = map[i].baryCoords[1];
        //const Real fz = map[i].baryCoords[2];
        //out[i] = in[cube[0]] * ((1-fx) * (1-fy) * (1-fz))
        //       + in[cube[1]] * ((  fx) * (1-fy) * (1-fz))
        //       + in[cube[2]] * ((1-fx) * (  fy) * (1-fz))
        //       + in[cube[3]] * ((  fx) * (  fy) * (1-fz))
        //       + in[cube[4]] * ((1-fx) * (1-fy) * (  fz))
        //       + in[cube[5]] * ((  fx) * (1-fy) * (  fz))
        //       + in[cube[6]] * ((1-fx) * (  fy) * (  fz))
        //       + in[cube[7]] * ((  fx) * (  fy) * (  fz));

        res = CudaVec3<float>::make(in [c.i          ]) * ((1-c.fx) * (1-c.fy) * (1-c.fz))
                + CudaVec3<float>::make(in [c.i+1        ]) * ((  c.fx) * (1-c.fy) * (1-c.fz))
                + CudaVec3<float>::make(in [c.i  +nx     ]) * ((1-c.fx) * (  c.fy) * (1-c.fz))
                + CudaVec3<float>::make(in [c.i+1+nx     ]) * ((  c.fx) * (  c.fy) * (1-c.fz))
                + CudaVec3<float>::make(in [c.i     +nxny]) * ((1-c.fx) * (1-c.fy) * (  c.fz))
                + CudaVec3<float>::make(in [c.i+1   +nxny]) * ((  c.fx) * (1-c.fy) * (  c.fz))
                + CudaVec3<float>::make(in [c.i  +nx+nxny]) * ((1-c.fx) * (  c.fy) * (  c.fz))
                + CudaVec3<float>::make(in [c.i+1+nx+nxny]) * ((  c.fx) * (  c.fy) * (  c.fz));
    }

    out[index] = CudaVec4<float>::make(res);
}

template<class TIn>
__global__ void RegularGridMapperCuda3f_applyJT_kernel(unsigned int size, unsigned int maxNOut, const GPULinearMap* mapT, float* out, const TIn* in)
{
    const int index0 = umul24(blockIdx.x,BSIZE); //blockDim.x;
    const int index1 = threadIdx.x;

    //! Dynamically allocated shared memory to reorder global memory access
    extern  __shared__  float temp[];

    CudaVec3<float> res = CudaVec3<float>::make(0,0,0);
    //res += *in * mapT[index0+index1].f;

    mapT+=umul24(index0,maxNOut)+index1;
    for (int s = 0; s < maxNOut; s++)
    {
        GPULinearMap data = *mapT;
        mapT+=BSIZE;
        if (data.i != -1)
            res += CudaVec3<float>::make(in[data.i]) * data.f;
    }

    const int index3 = umul24(index1,3);

    temp[index3  ] = res.x;
    temp[index3+1] = res.y;
    temp[index3+2] = res.z;

    __syncthreads();

    out += umul24(index0,3);
    out[index1        ] += temp[index1        ];
    out[index1+  BSIZE] += temp[index1+  BSIZE];
    out[index1+2*BSIZE] += temp[index1+2*BSIZE];
}

template<class TIn>
__global__ void RegularGridMapperCuda3f1_applyJT_kernel(unsigned int size, unsigned int maxNOut, const GPULinearMap* mapT, CudaVec4<float>* out, const TIn* in)
{
    const int index0 = umul24(blockIdx.x,BSIZE); //blockDim.x;
    const int index1 = threadIdx.x;
    const int index = index0+index1;

    CudaVec3<float> res = CudaVec3<float>::make(0,0,0);
    //res += *in * mapT[index0+index1].f;

    mapT+=umul24(index0,maxNOut)+index1;
    for (int s = 0; s < maxNOut; s++)
    {
        GPULinearMap data = *mapT;
        mapT+=BSIZE;
        if (data.i != -1)
            res += CudaVec3<float>::make(in [data.i]) * data.f;
    }

    CudaVec4<float> o = out[index];
    o.x += res.x;
    o.y += res.y;
    o.z += res.z;
    out[index] = o;
}





template<class TIn>
__global__ void MeshMapperCuda3f_apply_kernel(unsigned int size, unsigned int maxN, const GPULinearMap* map, float* out, const TIn* in)
{
    const int index0 = umul24(blockIdx.x,BSIZE); //blockDim.x;
    const int index1 = threadIdx.x;

    //! Dynamically allocated shared memory to reorder global memory access
    extern  __shared__  float temp[];

    CudaVec3<float> res = CudaVec3<float>::make(0,0,0);
    //res += *in * mapT[index0+index1].f;

    map+=umul24(index0,maxN)+index1;
    for (int s = 0; s < maxN; s++)
    {
        GPULinearMap data = *map;
        map+=BSIZE;
        if (data.i != 0)
            res += CudaVec3<float>::make(in[data.i-1]) * data.f;
    }

    const int index3 = umul24(index1,3);

    temp[index3  ] = res.x;
    temp[index3+1] = res.y;
    temp[index3+2] = res.z;

    __syncthreads();

    out += umul24(index0,3);
    out[index1        ] = temp[index1        ];
    out[index1+  BSIZE] = temp[index1+  BSIZE];
    out[index1+2*BSIZE] = temp[index1+2*BSIZE];
}

template<class TIn>
__global__ void MeshMapperCuda3f1_apply_kernel(unsigned int size, unsigned int maxN, const GPULinearMap* map, CudaVec4<float>* out, const TIn* in)
{
    const int index0 = umul24(blockIdx.x,BSIZE); //blockDim.x;
    const int index1 = threadIdx.x;
    const int index = index0+index1;

    CudaVec3<float> res = CudaVec3<float>::make(0,0,0);
    //res += *in * mapT[index0+index1].f;

    map+=umul24(index0,maxN)+index1;
    for (int s = 0; s < maxN; s++)
    {
        GPULinearMap data = *map;
        map+=BSIZE;
        if (data.i != 0)
            res += CudaVec3<float>::make(in[data.i-1]) * data.f;
    }

    CudaVec4<float> o;
    o.x = res.x;
    o.y = res.y;
    o.z = res.z;
    o.w = 0.0f;
    out[index] = o;
}






template<class TIn>
__global__ void MeshMapperCuda3f_applyPEq_kernel(unsigned int size, unsigned int maxN, const GPULinearMap* map, float* out, const TIn* in)
{
    const int index0 = umul24(blockIdx.x,BSIZE); //blockDim.x;
    const int index1 = threadIdx.x;

    //! Dynamically allocated shared memory to reorder global memory access
    extern  __shared__  float temp[];

    CudaVec3<float> res = CudaVec3<float>::make(0,0,0);
    //res += *in * mapT[index0+index1].f;

    map+=umul24(index0,maxN)+index1;
    for (int s = 0; s < maxN; s++)
    {
        GPULinearMap data = *map;
        map+=BSIZE;
        if (data.i != 0)
            res += CudaVec3<float>::make(in[data.i-1]) * data.f;
    }

    const int index3 = umul24(index1,3);

    temp[index3  ] = res.x;
    temp[index3+1] = res.y;
    temp[index3+2] = res.z;

    __syncthreads();

    out += umul24(index0,3);
    out[index1        ] += temp[index1        ];
    out[index1+  BSIZE] += temp[index1+  BSIZE];
    out[index1+2*BSIZE] += temp[index1+2*BSIZE];
}

template<class TIn>
__global__ void MeshMapperCuda3f1_applyPEq_kernel(unsigned int size, unsigned int maxN, const GPULinearMap* map, CudaVec4<float>* out, const TIn* in)
{
    const int index0 = umul24(blockIdx.x,BSIZE); //blockDim.x;
    const int index1 = threadIdx.x;
    const int index = index0+index1;

    CudaVec3<float> res = CudaVec3<float>::make(0,0,0);
    //res += *in * mapT[index0+index1].f;

    map+=umul24(index0,maxN)+index1;
    for (int s = 0; s < maxN; s++)
    {
        GPULinearMap data = *map;
        map+=BSIZE;
        if (data.i != 0)
            res += CudaVec3<float>::make(in[data.i-1]) * data.f;
    }

    CudaVec4<float> o = out[index];
    o.x += res.x;
    o.y += res.y;
    o.z += res.z;
    out[index] = o;
}

//////////////////////
// CPU-side methods //
//////////////////////

void RegularGridMapperCuda3f_apply(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    RegularGridMapperCuda3f_apply_kernel<CudaVec3<float> ><<< grid, threads, BSIZE*3*sizeof(float) >>>(size, gridsize[0], gridsize[0]*gridsize[1], (const GPUCubeData*)map, (float*)out, (const CudaVec3<float>*)in);
}

void RegularGridMapperCuda3f_applyJ(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    RegularGridMapperCuda3f_apply_kernel<CudaVec3<float> ><<< grid, threads, BSIZE*3*sizeof(float) >>>(size, gridsize[0], gridsize[0]*gridsize[1], (const GPUCubeData*)map, (float*)out, (const CudaVec3<float>*)in);
}

void RegularGridMapperCuda3f_applyJT(unsigned int insize, unsigned int maxNOut, const unsigned int* gridsize, const void* mapT, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((insize+BSIZE-1)/BSIZE,1);
    RegularGridMapperCuda3f_applyJT_kernel<CudaVec3<float> ><<< grid, threads, BSIZE*3*sizeof(float) >>>(insize, maxNOut, (const GPULinearMap*)mapT, (float*)out, (const CudaVec3<float>*)in);
}


void RegularGridMapperCuda3f1_apply(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    RegularGridMapperCuda3f1_apply_kernel<CudaVec4<float> ><<< grid, threads >>>(size, gridsize[0], gridsize[0]*gridsize[1], (const GPUCubeData*)map, (CudaVec4<float>*)out, (const CudaVec4<float>*)in);
}

void RegularGridMapperCuda3f1_applyJ(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    RegularGridMapperCuda3f1_apply_kernel<CudaVec4<float> ><<< grid, threads >>>(size, gridsize[0], gridsize[0]*gridsize[1], (const GPUCubeData*)map, (CudaVec4<float>*)out, (const CudaVec4<float>*)in);
}

void RegularGridMapperCuda3f1_applyJT(unsigned int insize, unsigned int maxNOut, const unsigned int* gridsize, const void* mapT, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((insize+BSIZE-1)/BSIZE,1);
    RegularGridMapperCuda3f1_applyJT_kernel<CudaVec4<float> ><<< grid, threads >>>(insize, maxNOut, (const GPULinearMap*)mapT, (CudaVec4<float>*)out, (const CudaVec4<float>*)in);
}


void RegularGridMapperCuda3f1_3f_apply(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    RegularGridMapperCuda3f_apply_kernel<CudaVec4<float> ><<< grid, threads, BSIZE*3*sizeof(float) >>>(size, gridsize[0], gridsize[0]*gridsize[1], (const GPUCubeData*)map, (float*)out, (const CudaVec4<float>*)in);
}

void RegularGridMapperCuda3f1_3f_applyJ(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    RegularGridMapperCuda3f_apply_kernel<CudaVec4<float> ><<< grid, threads, BSIZE*3*sizeof(float) >>>(size, gridsize[0], gridsize[0]*gridsize[1], (const GPUCubeData*)map, (float*)out, (const CudaVec4<float>*)in);
}

void RegularGridMapperCuda3f_3f1_applyJT(unsigned int insize, unsigned int maxNOut, const unsigned int* gridsize, const void* mapT, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((insize+BSIZE-1)/BSIZE,1);
    RegularGridMapperCuda3f_applyJT_kernel<CudaVec4<float> ><<< grid, threads, BSIZE*3*sizeof(float) >>>(insize, maxNOut, (const GPULinearMap*)mapT, (float*)out, (const CudaVec4<float>*)in);
}


void RegularGridMapperCuda3f_3f1_apply(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    RegularGridMapperCuda3f1_apply_kernel<CudaVec3<float> ><<< grid, threads >>>(size, gridsize[0], gridsize[0]*gridsize[1], (const GPUCubeData*)map, (CudaVec4<float>*)out, (const CudaVec3<float>*)in);
}

void RegularGridMapperCuda3f_3f1_applyJ(unsigned int size, const unsigned int* gridsize, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    RegularGridMapperCuda3f1_apply_kernel<CudaVec3<float> ><<< grid, threads >>>(size, gridsize[0], gridsize[0]*gridsize[1], (const GPUCubeData*)map, (CudaVec4<float>*)out, (const CudaVec3<float>*)in);
}

void RegularGridMapperCuda3f1_3f_applyJT(unsigned int insize, unsigned int maxNOut, const unsigned int* gridsize, const void* mapT, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((insize+BSIZE-1)/BSIZE,1);
    RegularGridMapperCuda3f1_applyJT_kernel<CudaVec3<float> ><<< grid, threads >>>(insize, maxNOut, (const GPULinearMap*)mapT, (CudaVec4<float>*)out, (const CudaVec3<float>*)in);
}



void MeshMapperCuda3f_apply(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MeshMapperCuda3f_apply_kernel<CudaVec3<float> ><<< grid, threads, BSIZE*3*sizeof(float) >>>(size, maxN, (const GPULinearMap*)map, (float*)out, (const CudaVec3<float>*)in);
}

void MeshMapperCuda3f1_apply(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MeshMapperCuda3f1_apply_kernel<CudaVec4<float> ><<< grid, threads >>>(size, maxN, (const GPULinearMap*)map, (CudaVec4<float>*)out, (const CudaVec4<float>*)in);
}

void MeshMapperCuda3f_3f1_apply(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MeshMapperCuda3f1_apply_kernel<CudaVec3<float> ><<< grid, threads >>>(size, maxN, (const GPULinearMap*)map, (CudaVec4<float>*)out, (const CudaVec3<float>*)in);
}

void MeshMapperCuda3f1_3f_apply(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MeshMapperCuda3f_apply_kernel<CudaVec4<float> ><<< grid, threads, BSIZE*3*sizeof(float) >>>(size, maxN, (const GPULinearMap*)map, (float*)out, (const CudaVec4<float>*)in);
}


void MeshMapperCuda3f_applyPEq(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MeshMapperCuda3f_applyPEq_kernel<CudaVec3<float> ><<< grid, threads, BSIZE*3*sizeof(float) >>>(size, maxN, (const GPULinearMap*)map, (float*)out, (const CudaVec3<float>*)in);
}

void MeshMapperCuda3f1_applyPEq(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MeshMapperCuda3f1_applyPEq_kernel<CudaVec4<float> ><<< grid, threads >>>(size, maxN, (const GPULinearMap*)map, (CudaVec4<float>*)out, (const CudaVec4<float>*)in);
}

void MeshMapperCuda3f_3f1_applyPEq(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MeshMapperCuda3f1_applyPEq_kernel<CudaVec3<float> ><<< grid, threads >>>(size, maxN, (const GPULinearMap*)map, (CudaVec4<float>*)out, (const CudaVec3<float>*)in);
}

void MeshMapperCuda3f1_3f_applyPEq(unsigned int size, unsigned int maxN, const void* map, void* out, const void* in)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MeshMapperCuda3f_applyPEq_kernel<CudaVec4<float> ><<< grid, threads, BSIZE*3*sizeof(float) >>>(size, maxN, (const GPULinearMap*)map, (float*)out, (const CudaVec4<float>*)in);
}

#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
