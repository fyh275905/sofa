#include "hip/hip_runtime.h"
#include "CudaCommon.h"
#include "CudaMath.h"
#include <stdio.h>
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION != 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    void CudaCollisionDetection_runTests(unsigned int nbTests, unsigned int maxPoints, const void* tests, void* nresults);
}

struct /*__align__(16)*/ GPUContact
{
    int p1;
    float3 p2;
    float distance;
    float3 normal;
};

struct GPUTest
{
    GPUContact* result;
    const float3* points;
    const float* radius;
    const float* grid;
    matrix3 rotation;
    float3 translation;
    float margin;
    int nbPoints;
    int gridnx, gridny, gridnz;
    float3 gridbbmin, gridbbmax;
    float3 gridp0, gridinvdp;
};

//////////////////////
// GPU-side methods //
//////////////////////

__shared__ GPUTest curTest;

__global__ void CudaCollisionDetection_runTests_kernel(const GPUTest* tests, int* nresults)
{
    if (threadIdx.x == 0)
        curTest = tests[blockIdx.x];


    __syncthreads();

    //! Dynamically allocated shared memory to compact results
    extern  __shared__  int scan[];

    float3 p;
    float distance;
    float3 grad = make_float3(0,0,0);
    //float3 normal;
    int n = 0;
    if (threadIdx.x < curTest.nbPoints)
    {
        p = curTest.points[threadIdx.x];
        p = curTest.rotation * p;
        p += curTest.translation;

        float3 coefs = mul(p-curTest.gridp0, curTest.gridinvdp);
        int x = __float2int_rd(coefs.x);
        int y = __float2int_rd(coefs.y);
        int z = __float2int_rd(coefs.z);
        if ((unsigned)x < curTest.gridnx-1
            && (unsigned)y < curTest.gridny-1
            && (unsigned)z < curTest.gridnz-1)
        {
            int nx = curTest.gridnx;
            int nxny = nx*curTest.gridny;
            coefs.x -= __int2float_rd(x);
            coefs.y -= __int2float_rd(y);
            coefs.z -= __int2float_rd(z);
            const float* gval = curTest.grid + (x+nx*y+nxny*z);
            float d000      = gval[0        ];
            float d100_d000 = gval[1        ] - d000;
            float d010      = gval[  nx     ];
            float d110_d010 = gval[1+nx     ] - d010;
            float d001      = gval[     nxny];
            float d101_d001 = gval[1   +nxny] - d001;
            float d011      = gval[  nx+nxny];
            float d111_d011 = gval[1+nx+nxny] - d011;
            float dx00      = d000 + (d100_d000)*coefs.x;
            float dx10_dx00 = d010 + (d110_d010)*coefs.x - dx00;
            float dx01      = d001 + (d101_d001)*coefs.x;
            float dx11_dx01 = d011 + (d111_d011)*coefs.x - dx01;
            float dy0       = dx00 + (dx10_dx00)*coefs.y;
            float dy1_dy0   = dx01 + (dx11_dx01)*coefs.y - dy0;
            distance = dy0 + (dy1_dy0)*coefs.z;
            float r = 0;
            if (curTest.radius)
            {
                r = curTest.radius[threadIdx.x];
            }
            if (distance < curTest.margin+r)
            {
                n = 1;
                grad.z = dy1_dy0;
                grad.y = (dx10_dx00) + ((dx11_dx01)-(dx10_dx00))*coefs.z;
                dy0     = d100_d000 + (d110_d010 - d100_d000)*coefs.y;
                dy1_dy0 = d101_d001 + (d111_d011 - d101_d001)*coefs.y - dy0;
                grad.x = dy0 + (dy1_dy0)*coefs.z;
                grad *= invnorm(grad);
                //normal = grad;
                p -= grad*distance;
                //distance -= r;
                distance = r;
                //grad = make_float3(0,1,1);
            }
        }
    }

    scan[threadIdx.x] = n;

    for (int i=1; i<curTest.nbPoints; i<<=1)
    {
        __syncthreads();
        if (threadIdx.x>=i)
            scan[threadIdx.x] = scan[threadIdx.x] + scan[threadIdx.x - i];
    }

    if (n)
    {
        GPUContact c;
        c.p1 = threadIdx.x;
        c.p2 = p;
        c.distance = distance;
        c.normal = -grad;
        //c.normal = normal; //make_float3(-grad.x,-grad.y,-grad.z); //-grad;
        curTest.result[scan[threadIdx.x]-1] = c;
        //curTest.result[scan[threadIdx.x]-1].p1 = threadIdx.x;
        //curTest.result[scan[threadIdx.x]-1].p2 = p;
        //curTest.result[scan[threadIdx.x]-1].distance = distance;
        //curTest.result[scan[threadIdx.x]-1].normal = normal;

    }
    if (threadIdx.x == curTest.nbPoints-1)
        nresults[blockIdx.x] = scan[curTest.nbPoints-1];
}

//////////////////////
// CPU-side methods //
//////////////////////

void CudaCollisionDetection_runTests(unsigned int nbTests, unsigned int maxPoints, const void* tests, void* nresults)
{
    printf("sizeof(GPUTest)=%d\nsizeof(GPUContact)=%d\nsizeof(matrix3)=%d\n",sizeof(GPUTest),sizeof(GPUContact),sizeof(matrix3));
    const GPUTest* gputests = (const GPUTest*)tests;
    // round up to 16
    //maxPoints = (maxPoints+15)&-16;
    dim3 threads(maxPoints,1);
    dim3 grid(nbTests,1);
    CudaCollisionDetection_runTests_kernel<<< grid, threads, threads.x*sizeof(int) >>>(gputests, (int*)nresults);

}

#if defined(__cplusplus) && CUDA_VERSION != 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
