#include "hip/hip_runtime.h"
#include "CudaCommon.h"
#include "CudaMath.h"
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION != 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    void RigidContactMapperCuda3f_setPoints2(unsigned int size, unsigned int nbTests, unsigned int maxPoints, const void* tests, const void* contacts, void* map);
    void SubsetContactMapperCuda3f_setPoints1(unsigned int size, unsigned int nbTests, unsigned int maxPoints, unsigned int nbPointsPerElem, const void* tests, const void* contacts, void* map);
}

struct /*__align__(16)*/ GPUContact
{
    int p1;
    float3 p2;
    float distance;
    float3 normal;
};

struct /*__align__(8)*/ GPUTestEntry
{
    int firstIndex;
    int curSize;
    int maxSize;
    int newIndex;
    int elem1,elem2;
};

//////////////////////
// GPU-side methods //
//////////////////////

__shared__ GPUTestEntry curTestEntry;

__global__ void RigidContactMapperCuda3f_setPoints2_kernel(const GPUTestEntry* tests, const GPUContact* contacts, float3* map)
{
    if (threadIdx.x == 0)
        curTestEntry = tests[blockIdx.x];

    __syncthreads();

    GPUContact c = contacts[curTestEntry.firstIndex + threadIdx.x];
    if (threadIdx.x < curTestEntry.curSize)
    {
        map[curTestEntry.newIndex + threadIdx.x] = c.p2;
    }
}

__global__ void SubsetContactMapperCuda3f_setPoints1_kernel(unsigned int nbPointsPerElem, const GPUTestEntry* tests, const GPUContact* contacts, int* map)
{
    if (threadIdx.x == 0)
        curTestEntry = tests[blockIdx.x];

    __syncthreads();

    GPUContact c = contacts[curTestEntry.firstIndex + threadIdx.x];
    if (threadIdx.x < curTestEntry.curSize)
    {
        map[curTestEntry.newIndex + threadIdx.x] = umul24(curTestEntry.elem1,nbPointsPerElem) + c.p1;
    }
}


//////////////////////
// CPU-side methods //
//////////////////////

void RigidContactMapperCuda3f_setPoints2(unsigned int size, unsigned int nbTests, unsigned int maxPoints, const void* tests, const void* contacts, void* map)
{
    // round up to 16
    //maxPoints = (maxPoints+15)&-16;
    dim3 threads(maxPoints,1);
    dim3 grid(nbTests,1);
    RigidContactMapperCuda3f_setPoints2_kernel<<< grid, threads >>>((const GPUTestEntry*)tests, (GPUContact*)contacts, (float3*)map);
}

void SubsetContactMapperCuda3f_setPoints1(unsigned int size, unsigned int nbTests, unsigned int maxPoints, unsigned int nbPointsPerElem, const void* tests, const void* contacts, void* map)
{
    // round up to 16
    //maxPoints = (maxPoints+15)&-16;
    dim3 threads(maxPoints,1);
    dim3 grid(nbTests,1);
    SubsetContactMapperCuda3f_setPoints1_kernel<<< grid, threads >>>(nbPointsPerElem, (const GPUTestEntry*)tests, (GPUContact*)contacts, (int*)map);

}

#if defined(__cplusplus) && CUDA_VERSION != 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
