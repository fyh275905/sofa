#include "hip/hip_runtime.h"
/******************************************************************************
*       SOFA, Simulation Open-Framework Architecture, version 1.0 beta 4      *
*                (c) 2006-2009 MGH, INRIA, USTL, UJF, CNRS                    *
*                                                                             *
* This library is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This library is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this library; if not, write to the Free Software Foundation,     *
* Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301 USA.          *
*******************************************************************************
*                               SOFA :: Modules                               *
*                                                                             *
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
/* PART OF THIS FILE IS FROM NVIDIA CUDA SDK particles demo:
 *
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */


#include "CudaCommon.h"
#include "CudaMath.h"
#include "hip/hip_runtime.h"
#include "radixsort.h"

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    void SpatialGridContainer3f_computeHash(int cellBits, float cellWidth, int nbPoints, void* particleIndex8, void* particleHash8, const void* x);
    void SpatialGridContainer3f1_computeHash(int cellBits, float cellWidth, int nbPoints, void* particleIndex8, void* particleHash8, const void* x);
    void SpatialGridContainer_findCellRange(int cellBits, float cellWidth, int nbPoints, const void* particleHash8, void* cellRange);
//void SpatialGridContainer3f_reorderData(int nbPoints, const void* particleHash, void* sorted, const void* x);
//void SpatialGridContainer3f1_reorderData(int nbPoints, const void* particleHash, void* sorted, const void* x);
}

#define USE_TEX  1
#define USE_SORT 1

struct GridParams
{
    float cellWidth;
    float invCellWidth;
    int cellMask;
    float halfCellWidth;
    float invHalfCellWidth;
};

// large prime numbers
#define HASH_PX 73856093
#define HASH_PY 19349663
#define HASH_PZ 83492791

//////////////////////
// GPU-side methods //
//////////////////////

#if USE_TEX
#if USE_SORT
texture<uint2, 1, hipReadModeElementType> particleHashTex;
texture<uint2, 1, hipReadModeElementType> cellRangeTex;
#else
texture<unsigned int, 1, hipReadModeElementType> gridCountersTex;
texture<unsigned int, 1, hipReadModeElementType> gridCellsTex;
#endif
#endif

__constant__ GridParams gridParams;

// calculate cell in grid from position
template<class T>
__device__ int3 calcGridPos(T p)
{
    int3 i;
    i.x = __float2int_rd(p.x * gridParams.invCellWidth);
    i.y = __float2int_rd(p.y * gridParams.invCellWidth);
    i.z = __float2int_rd(p.z * gridParams.invCellWidth);
    return i;
}

// calculate address in grid from position
__device__ unsigned int calcGridHashI(int3 p)
{
    //return ((p.x<<10)^(p.y<<5)^(p.z)) & gridParams.cellMask;
    //return ((p.x)^(p.y)^(p.z)) & gridParams.cellMask;
    return (__mul24(HASH_PX,p.x)^__mul24(HASH_PY,p.y)^__mul24(HASH_PZ,p.z)) & gridParams.cellMask;
    //return (p.x) & gridParams.cellMask;
}

// calculate address in grid from position
template<class T>
__device__ unsigned int calcGridHash(T p)
{
    return calcGridHashI(calcGridPos(p));
}


__device__ __inline__ float3 getPos3(const float4* pos, int index0, int index)
{
    float4 p = pos[index];
    return make_float3(p.x,p.y,p.z);
}

__shared__ float ftemp[BSIZE*3];

__device__ __inline__ float3 getPos3(const float3* pos, int index0, int index)
{
    //return pos[index];
    int index03 = __umul24(index0,3);
    int index3 = __umul24(threadIdx.x,3);
    ftemp[threadIdx.x] = ((const float*)pos)[index03+threadIdx.x];
    ftemp[threadIdx.x+BSIZE] = ((const float*)pos)[index03+threadIdx.x+BSIZE];
    ftemp[threadIdx.x+2*BSIZE] = ((const float*)pos)[index03+threadIdx.x+2*BSIZE];
    __syncthreads();
    return make_float3(ftemp[index3],ftemp[index3+1],ftemp[index3+2]);
}

__device__ __inline__ float4 getPos4(const float4* pos, int index0, int index)
{
    return pos[index];
}

__device__ __inline__ float4 getPos4(const float3* pos, int index0, int index)
{
    int index3 = __umul24(threadIdx.x,3);
    pos += index0;
    ftemp[threadIdx.x] = ((const float*)pos)[threadIdx.x];
    ftemp[threadIdx.x+BSIZE] = ((const float*)pos)[threadIdx.x+BSIZE];
    ftemp[threadIdx.x+2*BSIZE] = ((const float*)pos)[threadIdx.x+2*BSIZE];
    __syncthreads();
    return make_float4(ftemp[index3],ftemp[index3+1],ftemp[index3+2],0.0f);
}

__device__ __inline__ float4 getPos4(const float4* pos, int index)
{
    return pos[index];
}

__device__ __inline__ float4 getPos4(const float3* pos, int index)
{
    float3 p = pos[index];
    return make_float4(p.x,p.y,p.z,1.0f);
}

// calculate grid hash value for each particle
template<class TIn>
__global__ void
computeHashD(const TIn* pos,
        uint* particleIndex8, uint*  particleHash8, int n)
{
    int index0 = __mul24(blockIdx.x, blockDim.x);
    int index = index0 + threadIdx.x;

    float3 p = getPos3(pos,index0,index);

    int3 hgpos;
    hgpos.x = __float2int_rd(p.x * gridParams.invHalfCellWidth);
    hgpos.y = __float2int_rd(p.y * gridParams.invHalfCellWidth);
    hgpos.z = __float2int_rd(p.z * gridParams.invHalfCellWidth);
    int halfcell = (hgpos.x&1) + ((hgpos.y&1)<<1) + ((hgpos.z&1)<<2);
    // compute the first cell to be influenced by the particle
    hgpos.x = (hgpos.x-1) >> 1;
    hgpos.y = (hgpos.y-1) >> 1;
    hgpos.z = (hgpos.z-1) >> 1;

    __shared__ unsigned int hx[BSIZE];
    __shared__ unsigned int hy[BSIZE];
    __shared__ unsigned int hz[BSIZE];
    int x = threadIdx.x;

    hx[x] = (__mul24(HASH_PX,hgpos.x) << 3)+halfcell;
    hy[x] = __mul24(HASH_PY,hgpos.y);
    hz[x] = __mul24(HASH_PZ,hgpos.z);
    __syncthreads();
    uint3 dH;
    dH.x = (x&1 ? HASH_PX : 0);
    dH.y = (x&2 ? HASH_PY : 0);
    dH.z = (x&4 ? HASH_PZ : 0);
    int index0_8 = index0 << 3;
    for (unsigned int l = x; l < 8*BSIZE; l+=BSIZE)
    {
        particleIndex8[index0_8 + l] = index0 + (l>>3);
        uint3 h;
        h.x = hx[l>>3];
        h.y = hy[l>>3];
        h.z = hz[l>>3];
        int hc = h.x & 7;
        h.x = (h.x>>3) + dH.x;
        h.y += dH.y;
        h.z += dH.z;
        uint hash = ((h.x ^ h.y ^ h.z) & gridParams.cellMask)<<1;
        if (hc == (x&7)) ++hash;
        particleHash8[index0_8 + l] = hash;
    }
}

// find start of each cell in sorted particle list by comparing with previous hash value
// one thread per particle
__global__ void
findCellRangeD(const uint* particleHash,
        uint2 * cellRange, int n)
{
    unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    __shared__ uint hash[BSIZE];
    if (i < n)
    {
        hash[threadIdx.x] = particleHash[i]>>1;
        __syncthreads();
        bool first;
        if (i == 0) first = true;
        else
        {
            uint prev;
            if (threadIdx.x > 0)
                prev = hash[threadIdx.x-1];
            else
                prev = particleHash[i-1]>>1;
            first = (prev != hash[threadIdx.x]);
            if (first) // prev is the last of the previous cell
                cellRange[ prev ].y = i;
        }
        if (first)
            cellRange[ hash[threadIdx.x] ].x = i;
        if (i == n-1)
            cellRange[ hash[threadIdx.x] ].y = n;
    }
}

// rearrange particle data into sorted order
template<class TIn>
__global__ void
reorderDataD(const uint2*  particleHash,  // particle id sorted by hash
        const TIn* oldPos,
        float4* sortedPos, int n
            )
{
    int index0 = __mul24(blockIdx.x, blockDim.x);
    int index = index0 + threadIdx.x;
    if (index < n)
    {
        volatile uint2 sortedData = particleHash[index];
        //float4 pos = getPos4(oldPos,index0,index);
        float4 pos = getPos4(oldPos,sortedData.y);
        sortedPos[index] = pos;
    }
}


//////////////////////
// CPU-side methods //
//////////////////////

void SpatialGridContainer3f_computeHash(int cellBits, float cellWidth, int nbPoints, void* particleIndex8, void* particleHash8, const void* x)
{
    GridParams p;
    p.cellWidth = cellWidth;
    p.invCellWidth = 1.0f/cellWidth;
    p.cellMask = (1<<cellBits)-1;
    p.halfCellWidth = cellWidth*0.5f;
    p.invHalfCellWidth = 2.0f/cellWidth;
    hipMemcpyToSymbol(HIP_SYMBOL(gridParams), &p, sizeof(GridParams));

    // First compute hash of each particle
    {
        dim3 threads(BSIZE,1);
        dim3 grid((nbPoints+BSIZE-1)/BSIZE,1);
        computeHashD<float3><<< grid, threads >>>((const float3*)x, (uint*)particleIndex8, (uint*)particleHash8, nbPoints);
    }
}

void SpatialGridContainer3f1_computeHash(int cellBits, float cellWidth, int nbPoints, void* particleIndex8, void* particleHash8, const void* x)
{
    GridParams p;
    p.cellWidth = cellWidth;
    p.invCellWidth = 1.0f/cellWidth;
    p.cellMask = (1<<cellBits)-1;
    p.halfCellWidth = cellWidth*0.5f;
    p.invHalfCellWidth = 2.0f/cellWidth;
    hipMemcpyToSymbol(HIP_SYMBOL(gridParams), &p, sizeof(GridParams));

    // First compute hash of each particle
    {
        dim3 threads(BSIZE,1);
        dim3 grid((nbPoints+BSIZE-1)/BSIZE,1);
        computeHashD<float4><<< grid, threads >>>((const float4*)x, (uint*)particleIndex8, (uint*)particleHash8, nbPoints);
    }
}

void SpatialGridContainer_findCellRange(int cellBits, float cellWidth, int nbPoints, const void* particleHash8, void* cellRange)
{
    hipMemset(cellRange, -1, (1<<cellBits)*2*sizeof(int));

    // Then find the start of each cell
    {
        dim3 threads(BSIZE,1);
        dim3 grid((8*nbPoints+BSIZE-1)/BSIZE,1);
        findCellRangeD<<< grid, threads >>>((const unsigned int*)particleHash8, (uint2*)cellRange, 8*nbPoints);
    }
}
/*
void SpatialGridContainer3f_reorderData(int nbPoints, const void* particleHash, void* sorted, const void* x)
{
    dim3 threads(BSIZE,1);
    dim3 grid((nbPoints+BSIZE-1)/BSIZE,1);
    reorderDataD<float3><<< grid, threads >>>((const uint2*)particleHash, (const float3*)x, (float4*)sorted, nbPoints);
}

void SpatialGridContainer3f1_reorderData(int nbPoints, const void* particleHash, void* sorted, const void* x)
{
    dim3 threads(BSIZE,1);
    dim3 grid((nbPoints+BSIZE-1)/BSIZE,1);
    reorderDataD<float4><<< grid, threads >>>((const uint2*)particleHash, (const float4*)x, (float4*)sorted, nbPoints);
}
*/
#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
