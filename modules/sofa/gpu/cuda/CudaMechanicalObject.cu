#include "hip/hip_runtime.h"
#include "CudaCommon.h"
#include "CudaMath.h"
#include "mycuda.h"
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION != 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    void MechanicalObjectCudaVec3f_vAssign(unsigned int size, void* res, const void* a);
    void MechanicalObjectCudaVec3f_vClear(unsigned int size, void* res);
    void MechanicalObjectCudaVec3f_vMEq(unsigned int size, void* res, float f);
    void MechanicalObjectCudaVec3f_vEqBF(unsigned int size, void* res, const void* b, float f);
    void MechanicalObjectCudaVec3f_vPEq(unsigned int size, void* res, const void* a);
    void MechanicalObjectCudaVec3f_vPEqBF(unsigned int size, void* res, const void* b, float f);
    void MechanicalObjectCudaVec3f_vAdd(unsigned int size, void* res, const void* a, const void* b);
    void MechanicalObjectCudaVec3f_vOp(unsigned int size, void* res, const void* a, const void* b, float f);
    void MechanicalObjectCudaVec3f_vIntegrate(unsigned int size, const void* a, void* v, void* x, float f_v_v, float f_v_a, float f_x_x, float f_x_v);
    void MechanicalObjectCudaVec3f_vPEqBF2(unsigned int size, void* res1, const void* b1, float f1, void* res2, const void* b2, float f2);
    void MechanicalObjectCudaVec3f_vPEq4BF2(unsigned int size, void* res1, const void* b11, float f11, const void* b12, float f12, const void* b13, float f13, const void* b14, float f14,
            void* res2, const void* b21, float f21, const void* b22, float f22, const void* b23, float f23, const void* b24, float f24);
    void MechanicalObjectCudaVec3f_vOp2(unsigned int size, void* res1, const void* a1, const void* b1, float f1, void* res2, const void* a2, const void* b2, float f2);
    int MechanicalObjectCudaVec3f_vDotTmpSize(unsigned int size);
    void MechanicalObjectCudaVec3f_vDot(unsigned int size, float* res, const void* a, const void* b, void* tmp, float* cputmp);

    void MechanicalObjectCudaVec3f1_vAssign(unsigned int size, void* res, const void* a);
    void MechanicalObjectCudaVec3f1_vClear(unsigned int size, void* res);
    void MechanicalObjectCudaVec3f1_vMEq(unsigned int size, void* res, float f);
    void MechanicalObjectCudaVec3f1_vEqBF(unsigned int size, void* res, const void* b, float f);
    void MechanicalObjectCudaVec3f1_vPEq(unsigned int size, void* res, const void* a);
    void MechanicalObjectCudaVec3f1_vPEqBF(unsigned int size, void* res, const void* b, float f);
    void MechanicalObjectCudaVec3f1_vAdd(unsigned int size, void* res, const void* a, const void* b);
    void MechanicalObjectCudaVec3f1_vOp(unsigned int size, void* res, const void* a, const void* b, float f);
    void MechanicalObjectCudaVec3f1_vIntegrate(unsigned int size, const void* a, void* v, void* x, float f_v_v, float f_v_a, float f_x_x, float f_x_v);
    void MechanicalObjectCudaVec3f1_vPEqBF2(unsigned int size, void* res1, const void* b1, float f1, void* res2, const void* b2, float f2);
    void MechanicalObjectCudaVec3f1_vPEq4BF2(unsigned int size, void* res1, const void* b11, float f11, const void* b12, float f12, const void* b13, float f13, const void* b14, float f14,
            void* res2, const void* b21, float f21, const void* b22, float f22, const void* b23, float f23, const void* b24, float f24);
    void MechanicalObjectCudaVec3f1_vOp2(unsigned int size, void* res1, const void* a1, const void* b1, float f1, void* res2, const void* a2, const void* b2, float f2);
    int MechanicalObjectCudaVec3f1_vDotTmpSize(unsigned int size);
    void MechanicalObjectCudaVec3f1_vDot(unsigned int size, float* res, const void* a, const void* b, void* tmp, float* cputmp);
}

//////////////////////
// GPU-side methods //
//////////////////////

__global__ void MechanicalObjectCudaVec1f_vClear_kernel(int size, float* res)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        res[index] = 0.0f;
    }
}

__global__ void MechanicalObjectCudaVec3f_vClear_kernel(int size, float3* res)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        res[index] = make_float3(0.0f,0.0f,0.0f);
    }
}

__global__ void MechanicalObjectCudaVec3f1_vClear_kernel(int size, float4* res)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        res[index] = make_float4(0.0f,0.0f,0.0f,0.0f);
    }
}

__global__ void MechanicalObjectCudaVec1f_vMEq_kernel(int size, float* res, float f)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        res[index] *= f;
    }
}

__global__ void MechanicalObjectCudaVec3f_vMEq_kernel(int size, float* res, float f)
{
    int index = umul24(blockIdx.x,BSIZE*3)+threadIdx.x;
    //if (index < size)
    {
        res[index] *= f;
        index += BSIZE;
        res[index] *= f;
        index += BSIZE;
        res[index] *= f;
        //float3 ri = res[index];
        //ri *= f;
        //res[index] = ri;
    }
}

__global__ void MechanicalObjectCudaVec3f1_vMEq_kernel(int size, float4* res, float f)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        //res[index] = res[index]*f;
        float4 v = res[index];
        v.x *= f;
        v.y *= f;
        v.z *= f;
        res[index] = v;
    }
}

__global__ void MechanicalObjectCudaVec1f_vEqBF_kernel(int size, float* res, const float* b, float f)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        res[index] = b[index] * f;
    }
}

__global__ void MechanicalObjectCudaVec3f_vEqBF_kernel(int size, float* res, const float* b, float f)
{
    int index = umul24(blockIdx.x,BSIZE*3)+threadIdx.x;
    //if (index < size)
    {
        res[index] = b[index] * f;
        index += BSIZE;
        res[index] = b[index] * f;
        index += BSIZE;
        res[index] = b[index] * f;
        //float3 bi = b[index];
        //float3 ri = bi * f;
        //res[index] = ri;
    }
}

__global__ void MechanicalObjectCudaVec3f1_vEqBF_kernel(int size, float4* res, const float4* b, float f)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        //res[index] = b[index] * f;
        float4 v = b[index];
        v.x *= f;
        v.y *= f;
        v.z *= f;
        res[index] = v;
    }
}

__global__ void MechanicalObjectCudaVec1f_vPEq_kernel(int size, float* res, const float* a)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        res[index] += a[index];
    }
}

__global__ void MechanicalObjectCudaVec3f_vPEq_kernel(int size, float* res, const float* a)
{
    int index = umul24(blockIdx.x,BSIZE*3)+threadIdx.x;
    //if (index < size)
    {
        res[index] += a[index];
        index += BSIZE;
        res[index] += a[index];
        index += BSIZE;
        res[index] += a[index];
        //float3 ai = a[index];
        //float3 ri = res[index];
        //ri += ai;
        //res[index] = ri;
    }
}

__global__ void MechanicalObjectCudaVec3f1_vPEq_kernel(int size, float4* res, const float4* a)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        //res[index] += a[index];
        float4 v = res[index];
        float4 v2 = a[index];
        v.x += v2.x;
        v.y += v2.y;
        v.z += v2.z;
        res[index] = v;
    }
}

__global__ void MechanicalObjectCudaVec1f_vPEqBF_kernel(int size, float* res, const float* b, float f)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        res[index] += b[index] * f;
    }
}

__global__ void MechanicalObjectCudaVec3f_vPEqBF_kernel(int size, float* res, const float* b, float f)
{
    int index = umul24(blockIdx.x,BSIZE*3)+threadIdx.x;
    //if (index < size)
    {
        res[index] += b[index] * f;
        index += BSIZE;
        res[index] += b[index] * f;
        index += BSIZE;
        res[index] += b[index] * f;
        //float3 bi = b[index];
        //float3 ri = res[index];
        //ri += bi * f;
        //res[index] = ri;
    }
}

__global__ void MechanicalObjectCudaVec3f1_vPEqBF_kernel(int size, float4* res, const float4* b, float f)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        //res[index] += b[index] * f;
        float4 v = res[index];
        float4 v2 = b[index];
        v.x += v2.x*f;
        v.y += v2.y*f;
        v.z += v2.z*f;
        res[index] = v;
    }
}

__global__ void MechanicalObjectCudaVec1f_vPEqBF2_kernel(int size, float* res1, const float* b1, float f1, float* res2, const float* b2, float f2)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        res1[index] += b1[index] * f1;
        res2[index] += b2[index] * f2;
    }
}

__global__ void MechanicalObjectCudaVec3f_vPEqBF2_kernel(int size, float* res1, const float* b1, float f1, float* res2, const float* b2, float f2)
{
    int index = umul24(blockIdx.x,BSIZE*3)+threadIdx.x;
    //if (index < size)
    {
        res1[index] += b1[index] * f1;
        res2[index] += b2[index] * f2;
        index += BSIZE;
        res1[index] += b1[index] * f1;
        res2[index] += b2[index] * f2;
        index += BSIZE;
        res1[index] += b1[index] * f1;
        res2[index] += b2[index] * f2;
        //float3 bi = b[index];
        //float3 ri = res[index];
        //ri += bi * f;
        //res[index] = ri;
    }
}

__global__ void MechanicalObjectCudaVec3f1_vPEqBF2_kernel(int size, float4* res1, const float4* b1, float f1, float4* res2, const float4* b2, float f2)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        //res[index] += b[index] * f;
        float4 v = res1[index];
        float4 v2 = b1[index];
        v.x += v2.x*f1;
        v.y += v2.y*f1;
        v.z += v2.z*f1;
        res1[index] = v;
        v = res2[index];
        v2 = b2[index];
        v.x += v2.x*f2;
        v.y += v2.y*f2;
        v.z += v2.z*f2;
        res2[index] = v;
    }
}

__global__ void MechanicalObjectCudaVec1f_vPEq4BF2_kernel(int size, float* res1, const float* b11, float f11, const float* b12, float f12, const float* b13, float f13, const float* b14, float f14,
        float* res2, const float* b21, float f21, const float* b22, float f22, const float* b23, float f23, const float* b24, float f24)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        float r1,r2;
        r1 = res1[index];
        r2 = res2[index];
        r1 += b11[index] * f11;
        r2 += b21[index] * f21;
        r1 += b12[index] * f12;
        r2 += b22[index] * f22;
        r1 += b13[index] * f13;
        r2 += b23[index] * f23;
        r1 += b14[index] * f14;
        r2 += b24[index] * f24;
        res1[index] = r1;
        res2[index] = r2;
    }
}

__global__ void MechanicalObjectCudaVec3f_vPEq4BF2_kernel(int size, float* res1, const float* b11, float f11, const float* b12, float f12, const float* b13, float f13, const float* b14, float f14,
        float* res2, const float* b21, float f21, const float* b22, float f22, const float* b23, float f23, const float* b24, float f24)
{
    int index = umul24(blockIdx.x,BSIZE*3)+threadIdx.x;
    //if (index < size)
    {
        float r1,r2;
        r1 = res1[index];
        r2 = res2[index];
        r1 += b11[index] * f11;
        r2 += b21[index] * f21;
        r1 += b12[index] * f12;
        r2 += b22[index] * f22;
        r1 += b13[index] * f13;
        r2 += b23[index] * f23;
        r1 += b14[index] * f14;
        r2 += b24[index] * f24;
        res1[index] = r1;
        res2[index] = r2;
        index += BSIZE;
        r1 = res1[index];
        r2 = res2[index];
        r1 += b11[index] * f11;
        r2 += b21[index] * f21;
        r1 += b12[index] * f12;
        r2 += b22[index] * f22;
        r1 += b13[index] * f13;
        r2 += b23[index] * f23;
        r1 += b14[index] * f14;
        r2 += b24[index] * f24;
        res1[index] = r1;
        res2[index] = r2;
        index += BSIZE;
        r1 = res1[index];
        r2 = res2[index];
        r1 += b11[index] * f11;
        r2 += b21[index] * f21;
        r1 += b12[index] * f12;
        r2 += b22[index] * f22;
        r1 += b13[index] * f13;
        r2 += b23[index] * f23;
        r1 += b14[index] * f14;
        r2 += b24[index] * f24;
        res1[index] = r1;
        res2[index] = r2;
    }
}

__global__ void MechanicalObjectCudaVec3f1_vPEq4BF2_kernel(int size, float4* res1, const float4* b11, float f11, const float4* b12, float f12, const float4* b13, float f13, const float4* b14, float f14,
        float4* res2, const float4* b21, float f21, const float4* b22, float f22, const float4* b23, float f23, const float4* b24, float f24)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        float4 v = res1[index];
        float4 v2 = b11[index];
        v.x += v2.x*f11;
        v.y += v2.y*f11;
        v.z += v2.z*f11;
        v2 = b12[index];
        v.x += v2.x*f12;
        v.y += v2.y*f12;
        v.z += v2.z*f12;
        v2 = b13[index];
        v.x += v2.x*f13;
        v.y += v2.y*f13;
        v.z += v2.z*f13;
        v2 = b14[index];
        v.x += v2.x*f14;
        v.y += v2.y*f14;
        v.z += v2.z*f14;
        res1[index] = v;
        v = res2[index];
        v2 = b21[index];
        v.x += v2.x*f21;
        v.y += v2.y*f21;
        v.z += v2.z*f21;
        v2 = b22[index];
        v.x += v2.x*f22;
        v.y += v2.y*f22;
        v.z += v2.z*f22;
        v2 = b23[index];
        v.x += v2.x*f23;
        v.y += v2.y*f23;
        v.z += v2.z*f23;
        v2 = b24[index];
        v.x += v2.x*f24;
        v.y += v2.y*f24;
        v.z += v2.z*f24;
        res2[index] = v;
    }
}

__global__ void MechanicalObjectCudaVec1f_vAdd_kernel(int size, float* res, const float* a, const float* b)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        res[index] = a[index] + b[index];
    }
}

__global__ void MechanicalObjectCudaVec3f_vAdd_kernel(int size, float* res, const float* a, const float* b)
{
    int index = umul24(blockIdx.x,BSIZE*3)+threadIdx.x;
    //if (index < size)
    {
        res[index] = a[index] + b[index];
        index += BSIZE;
        res[index] = a[index] + b[index];
        index += BSIZE;
        res[index] = a[index] + b[index];
        //float3 ai = a[index];
        //float3 bi = b[index];
        //float3 ri = ai + bi;
        //res[index] = ri;
    }
}

__global__ void MechanicalObjectCudaVec3f1_vAdd_kernel(int size, float4* res, const float4* a, const float4* b)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        //res[index] = a[index] + b[index];
        float4 v = a[index];
        float4 v2 = b[index];
        v.x += v2.x;
        v.y += v2.y;
        v.z += v2.z;
        res[index] = v;
    }
}

__global__ void MechanicalObjectCudaVec1f_vOp_kernel(int size, float* res, const float* a, const float* b, float f)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        res[index] = a[index] + b[index] * f;
    }
}

__global__ void MechanicalObjectCudaVec3f_vOp_kernel(int size, float* res, const float* a, const float* b, float f)
{
    int index = umul24(blockIdx.x,BSIZE*3)+threadIdx.x;
    //if (index < size)
    {
        res[index] = a[index] + b[index] * f;
        index += BSIZE;
        res[index] = a[index] + b[index] * f;
        index += BSIZE;
        res[index] = a[index] + b[index] * f;
        //float3 ai = a[index];
        //float3 bi = b[index];
        //float3 ri = ai + bi * f;
        //res[index] = ri;
    }
}

__global__ void MechanicalObjectCudaVec3f1_vOp_kernel(int size, float4* res, const float4* a, const float4* b, float f)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        //res[index] = a[index] + b[index] * f;
        float4 v = a[index];
        float4 v2 = b[index];
        v.x += v2.x*f;
        v.y += v2.y*f;
        v.z += v2.z*f;
        res[index] = v;
    }
}


__global__ void MechanicalObjectCudaVec1f_vOp2_kernel(int size, float* res1, const float* a1, const float* b1, float f1, float* res2, const float* a2, const float* b2, float f2)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        res1[index] = a1[index] + b1[index] * f1;
        res2[index] = a2[index] + b2[index] * f2;
    }
}

__global__ void MechanicalObjectCudaVec3f_vOp2_kernel(int size, float* res1, const float* a1, const float* b1, float f1, float* res2, const float* a2, const float* b2, float f2)
{
    int index = umul24(blockIdx.x,BSIZE*3)+threadIdx.x;
    //if (index < size)
    {
        res1[index] = a1[index] + b1[index] * f1;
        res2[index] = a2[index] + b2[index] * f2;
        index += BSIZE;
        res1[index] = a1[index] + b1[index] * f1;
        res2[index] = a2[index] + b2[index] * f2;
        index += BSIZE;
        res1[index] = a1[index] + b1[index] * f1;
        res2[index] = a2[index] + b2[index] * f2;
    }
}

__global__ void MechanicalObjectCudaVec3f1_vOp2_kernel(int size, float4* res1, const float4* a1, const float4* b1, float f1, float4* res2, const float4* a2, const float4* b2, float f2)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        //res[index] = a[index] + b[index] * f;
        float4 v = a1[index];
        float4 v2 = b1[index];
        v.x += v2.x*f1;
        v.y += v2.y*f1;
        v.z += v2.z*f1;
        res1[index] = v;
        v = a2[index];
        v2 = b2[index];
        v.x += v2.x*f2;
        v.y += v2.y*f2;
        v.z += v2.z*f2;
        res2[index] = v;
    }
}

__global__ void MechanicalObjectCudaVec1f_vIntegrate_kernel(int size, const float* a, float* v, float* x, float f_v_v, float f_v_a, float f_x_x, float f_x_v)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        float vi = v[index]*f_v_v + a[index] * f_v_a;
        v[index] = vi;
        x[index] = x[index]*f_x_x + vi * f_x_v;
    }
}

__global__ void MechanicalObjectCudaVec3f_vIntegrate_kernel(int size, const float* a, float* v, float* x, float f_v_v, float f_v_a, float f_x_x, float f_x_v)
{
    int index = umul24(blockIdx.x,BSIZE*3)+threadIdx.x;
    //if (index < size)
    {
        float vi;
        vi = v[index]*f_v_v + a[index] * f_v_a;
        v[index] = vi;
        x[index] = x[index]*f_x_x + vi * f_x_v;
        index += BSIZE;
        vi = v[index]*f_v_v + a[index] * f_v_a;
        v[index] = vi;
        x[index] = x[index]*f_x_x + vi * f_x_v;
        index += BSIZE;
        vi = v[index]*f_v_v + a[index] * f_v_a;
        v[index] = vi;
        x[index] = x[index]*f_x_x + vi * f_x_v;
    }
}

__global__ void MechanicalObjectCudaVec3f1_vIntegrate_kernel(int size, const float4* a, float4* v, float4* x, float f_v_v, float f_v_a, float f_x_x, float f_x_v)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    //if (index < size)
    {
        //res[index] = a[index] + b[index] * f;
        float4 ai = a[index];
        float4 vi = v[index];
        float4 xi = x[index];
        vi.x = vi.x*f_v_v + ai.x*f_v_a;
        vi.y = vi.y*f_v_v + ai.y*f_v_a;
        vi.z = vi.z*f_v_v + ai.z*f_v_a;
        xi.x = xi.x*f_x_x + vi.x*f_x_v;
        xi.y = xi.y*f_x_x + vi.y*f_x_v;
        xi.z = xi.z*f_x_x + vi.z*f_x_v;
        v[index] = vi;
        x[index] = xi;
    }
}

#define RED_BSIZE 128
#define blockSize RED_BSIZE
//template<unsigned int blockSize>
__global__ void MechanicalObjectCudaVecf_vDot_kernel(unsigned int n, float* res, const float* a, const float* b)
{
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;

    unsigned int i = blockIdx.x*(blockSize) + tid;
    unsigned int gridSize = gridDim.x*(blockSize);
    sdata[tid] = 0;
    while (i < n) { sdata[tid] += a[i] * b[i]; i += gridSize; }
    __syncthreads();
#if blockSize >= 512
    //if (blockSize >= 512)
    {
        if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
    }
#endif
#if blockSize >= 256
    //if (blockSize >= 256)
    {
        if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
    }
#endif
#if blockSize >= 128
    //if (blockSize >= 128)
    {
        if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();
    }
#endif
    if (tid < 32)
    {
#if blockSize >= 64
        //if (blockSize >= 64)
        sdata[tid] += sdata[tid + 32];
#endif
#if blockSize >= 32
        //if (blockSize >= 32)
        sdata[tid] += sdata[tid + 16];
#endif
#if blockSize >= 16
        //if (blockSize >= 16)
        sdata[tid] += sdata[tid + 8];
#endif
#if blockSize >= 8
        //if (blockSize >= 8)
        sdata[tid] += sdata[tid + 4];
#endif
#if blockSize >= 4
        //if (blockSize >= 4)
        sdata[tid] += sdata[tid + 2];
#endif
#if blockSize >= 2
        //if (blockSize >= 2)
        sdata[tid] += sdata[tid + 1];
#endif
    }
    if (tid == 0) res[blockIdx.x] = sdata[0];
}

//template<unsigned int blockSize>
__global__ void MechanicalObjectCudaVecf_vSum_kernel(int n, float* res, const float* a)
{
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;

    unsigned int i = blockIdx.x*(blockSize) + tid;
    unsigned int gridSize = blockSize*gridDim.x;
    sdata[tid] = 0;
    while (i < n) { sdata[tid] += a[i]; i += gridSize; }
    __syncthreads();
#if blockSize >= 512
    //if (blockSize >= 512)
    {
        if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
    }
#endif
#if blockSize >= 256
    //if (blockSize >= 256)
    {
        if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
    }
#endif
#if blockSize >= 128
    //if (blockSize >= 128)
    {
        if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();
    }
#endif
    if (tid < 32)
    {
#if blockSize >= 64
        //if (blockSize >= 64)
        sdata[tid] += sdata[tid + 32];
#endif
#if blockSize >= 32
        //if (blockSize >= 32)
        sdata[tid] += sdata[tid + 16];
#endif
#if blockSize >= 16
        //if (blockSize >= 16)
        sdata[tid] += sdata[tid + 8];
#endif
#if blockSize >= 8
        //if (blockSize >= 8)
        sdata[tid] += sdata[tid + 4];
#endif
#if blockSize >= 4
        //if (blockSize >= 4)
        sdata[tid] += sdata[tid + 2];
#endif
#if blockSize >= 2
        //if (blockSize >= 2)
        sdata[tid] += sdata[tid + 1];
#endif
    }
    if (tid == 0) res[blockIdx.x] = sdata[0];
}

#undef blockSize

//////////////////////
// CPU-side methods //
//////////////////////

void MechanicalObjectCudaVec3f_vAssign(unsigned int size, void* res, const void* a)
{
    //dim3 threads(BSIZE,1);
    //dim3 grid((size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec3f_vAssign_kernel<<< grid, threads >>>(res, a);
    hipMemcpy(res, a, size*3*sizeof(float), hipMemcpyDeviceToDevice);
}

void MechanicalObjectCudaVec3f1_vAssign(unsigned int size, void* res, const void* a)
{
    //dim3 threads(BSIZE,1);
    //dim3 grid((size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec3f1_vAssign_kernel<<< grid, threads >>>(res, a);
    hipMemcpy(res, a, size*4*sizeof(float), hipMemcpyDeviceToDevice);
}

void MechanicalObjectCudaVec3f_vClear(unsigned int size, void* res)
{
    dim3 threads(BSIZE,1);
    //dim3 grid((size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec3f_vClear_kernel<<< grid, threads >>>(size, (float3*)res);
    //dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vClear_kernel<<< grid, threads >>>(3*size, (float*)res);
    hipMemset(res, 0, size*3*sizeof(float));
}

void MechanicalObjectCudaVec3f1_vClear(unsigned int size, void* res)
{
    dim3 threads(BSIZE,1);
    //dim3 grid((size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec3f1_vClear_kernel<<< grid, threads >>>(size, (float4*)res);
    //dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vClear_kernel<<< grid, threads >>>(4*size, (float*)res);
    hipMemset(res, 0, size*4*sizeof(float));
}

void MechanicalObjectCudaVec3f_vMEq(unsigned int size, void* res, float f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f_vMEq_kernel<<< grid, threads >>>(size, (float*)res, f);
    //dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vMEq_kernel<<< grid, threads >>>(3*size, (float*)res, f);
}

void MechanicalObjectCudaVec3f1_vMEq(unsigned int size, void* res, float f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f1_vMEq_kernel<<< grid, threads >>>(size, (float4*)res, f);
    //dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vMEq_kernel<<< grid, threads >>>(4*size, (float*)res, f);
}

void MechanicalObjectCudaVec3f_vEqBF(unsigned int size, void* res, const void* b, float f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f_vEqBF_kernel<<< grid, threads >>>(size, (float*)res, (const float*)b, f);
    //dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vEqBF_kernel<<< grid, threads >>>(3*size, (float*)res, (const float*)b, f);
}

void MechanicalObjectCudaVec3f1_vEqBF(unsigned int size, void* res, const void* b, float f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f1_vEqBF_kernel<<< grid, threads >>>(size, (float4*)res, (const float4*)b, f);
    //dim3 grid((4*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vEqBF_kernel<<< grid, threads >>>(4*size, (float*)res, (const float*)b, f);
}

void MechanicalObjectCudaVec3f_vPEq(unsigned int size, void* res, const void* a)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f_vPEq_kernel<<< grid, threads >>>(size, (float*)res, (const float*)a);
    //dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vPEq_kernel<<< grid, threads >>>(3*size, (float*)res, (const float*)a);
}

void MechanicalObjectCudaVec3f1_vPEq(unsigned int size, void* res, const void* a)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f1_vPEq_kernel<<< grid, threads >>>(size, (float4*)res, (const float4*)a);
    //dim3 grid((4*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vPEq_kernel<<< grid, threads >>>(4*size, (float*)res, (const float*)a);
}

void MechanicalObjectCudaVec3f_vPEqBF(unsigned int size, void* res, const void* b, float f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f_vPEqBF_kernel<<< grid, threads >>>(size, (float*)res, (const float*)b, f);
    //dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vPEqBF_kernel<<< grid, threads >>>(3*size, (float*)res, (const float*)b, f);
}

void MechanicalObjectCudaVec3f1_vPEqBF(unsigned int size, void* res, const void* b, float f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f1_vPEqBF_kernel<<< grid, threads >>>(size, (float4*)res, (const float4*)b, f);
    //dim3 grid((4*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vPEqBF_kernel<<< grid, threads >>>(4*size, (float*)res, (const float*)b, f);
}

void MechanicalObjectCudaVec3f_vPEqBF2(unsigned int size, void* res1, const void* b1, float f1, void* res2, const void* b2, float f2)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f_vPEqBF2_kernel<<< grid, threads >>>(size, (float*)res1, (const float*)b1, f1, (float*)res2, (const float*)b2, f2);
}

void MechanicalObjectCudaVec3f1_vPEqBF2(unsigned int size, void* res1, const void* b1, float f1, void* res2, const void* b2, float f2)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f1_vPEqBF2_kernel<<< grid, threads >>>(size, (float4*)res1, (const float4*)b1, f1, (float4*)res2, (const float4*)b2, f2);
}

void MechanicalObjectCudaVec3f_vPEq4BF2(unsigned int size, void* res1, const void* b11, float f11, const void* b12, float f12, const void* b13, float f13, const void* b14, float f14,
        void* res2, const void* b21, float f21, const void* b22, float f22, const void* b23, float f23, const void* b24, float f24)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f_vPEq4BF2_kernel<<< grid, threads >>>(size, (float*)res1, (const float*)b11, f11, (const float*)b12, f12, (const float*)b13, f13, (const float*)b14, f14,
            (float*)res2, (const float*)b21, f21, (const float*)b22, f22, (const float*)b23, f23, (const float*)b24, f24);
}

void MechanicalObjectCudaVec3f1_vPEq4BF2(unsigned int size, void* res1, const void* b11, float f11, const void* b12, float f12, const void* b13, float f13, const void* b14, float f14,
        void* res2, const void* b21, float f21, const void* b22, float f22, const void* b23, float f23, const void* b24, float f24)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f1_vPEq4BF2_kernel<<< grid, threads >>>(size, (float4*)res1, (const float4*)b11, f11, (const float4*)b12, f12, (const float4*)b13, f13, (const float4*)b14, f14,
            (float4*)res2, (const float4*)b21, f21, (const float4*)b22, f22, (const float4*)b23, f23, (const float4*)b24, f24);
}

void MechanicalObjectCudaVec3f_vOp2(unsigned int size, void* res1, const void* a1, const void* b1, float f1, void* res2, const void* a2, const void* b2, float f2)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f_vOp2_kernel<<< grid, threads >>>(size, (float*)res1, (const float*)a1, (const float*)b1, f1, (float*)res2, (const float*)a2, (const float*)b2, f2);
}

void MechanicalObjectCudaVec3f1_vOp2(unsigned int size, void* res1, const void* a1, const void* b1, float f1, void* res2, const void* a2, const void* b2, float f2)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f1_vOp2_kernel<<< grid, threads >>>(size, (float4*)res1, (const float4*)a1, (const float4*)b1, f1, (float4*)res2, (const float4*)a2, (const float4*)b2, f2);
}

void MechanicalObjectCudaVec3f_vAdd(unsigned int size, void* res, const void* a, const void* b)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f_vAdd_kernel<<< grid, threads >>>(size, (float*)res, (const float*)a, (const float*)b);
    //dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vAdd_kernel<<< grid, threads >>>(3*size, (float*)res, (const float*)a, (const float*)b);
}

void MechanicalObjectCudaVec3f1_vAdd(unsigned int size, void* res, const void* a, const void* b)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f1_vAdd_kernel<<< grid, threads >>>(size, (float4*)res, (const float4*)a, (const float4*)b);
    //dim3 grid((4*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vAdd_kernel<<< grid, threads >>>(4*size, (float*)res, (const float*)a, (const float*)b);
}

void MechanicalObjectCudaVec3f_vOp(unsigned int size, void* res, const void* a, const void* b, float f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f_vOp_kernel<<< grid, threads >>>(size, (float*)res, (const float*)a, (const float*)b, f);
    //dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vOp_kernel<<< grid, threads >>>(3*size, (float*)res, (const float*)a, (const float*)b, f);
}

void MechanicalObjectCudaVec3f1_vOp(unsigned int size, void* res, const void* a, const void* b, float f)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f1_vOp_kernel<<< grid, threads >>>(size, (float4*)res, (const float4*)a, (const float4*)b, f);
    //dim3 grid((4*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vOp_kernel<<< grid, threads >>>(4*size, (float*)res, (const float*)a, (const float*)b, f);
}

void MechanicalObjectCudaVec3f_vIntegrate(unsigned int size, const void* a, void* v, void* x, float f_v_v, float f_v_a, float f_x_x, float f_x_v)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f_vIntegrate_kernel<<< grid, threads >>>(size, (const float*)a, (float*)v, (float*)x, f_v_v, f_v_a, f_x_x, f_x_v);
    //dim3 grid((3*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vIntegrate_kernel<<< grid, threads >>>(3*size, (const float*)a, (float*)v, (float*)x, f_v_v, f_v_a, f_x_x, f_x_v);
}

void MechanicalObjectCudaVec3f1_vIntegrate(unsigned int size, const void* a, void* v, void* x, float f_v_v, float f_v_a, float f_x_x, float f_x_v)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    MechanicalObjectCudaVec3f1_vIntegrate_kernel<<< grid, threads >>>(size, (const float4*)a, (float4*)v, (float4*)x, f_v_v, f_v_a, f_x_x, f_x_v);
    //dim3 grid((4*size+BSIZE-1)/BSIZE,1);
    //MechanicalObjectCudaVec1f_vIntegrate_kernel<<< grid, threads >>>(4*size, (const float*)a, (float*)v, (float*)x, f_v_v, f_v_a, f_x_x, f_x_v);
}


int MechanicalObjectCudaVec3f_vDotTmpSize(unsigned int size)
{
    size *= 3;
    int nblocs = (size+RED_BSIZE-1)/RED_BSIZE;
    if (nblocs > 256) nblocs = 256;
    return nblocs;
}

void MechanicalObjectCudaVec3f_vDot(unsigned int size, float* res, const void* a, const void* b, void* tmp, float* rtmp)
{
    size *= 3;
    if (size==0)
    {
        *res = 0.0f;
    }
    else
    {
        int nblocs = (size+RED_BSIZE-1)/RED_BSIZE;
        if (nblocs > 256) nblocs = 256;
        dim3 threads(RED_BSIZE,1);
        dim3 grid(nblocs,1);
        //myprintf("size=%d, blocs=%dx%d\n",size,nblocs,RED_BSIZE);
        MechanicalObjectCudaVecf_vDot_kernel<<< grid, threads, RED_BSIZE * sizeof(float) >>>(size, (float*)tmp, (const float*)a, (const float*)b);
        if (nblocs == 1)
        {
            hipMemcpy(res,tmp,sizeof(float),hipMemcpyDeviceToHost);
        }
        else
        {
            /*
            dim3 threads(RED_BSIZE,1);
            dim3 grid(1,1);
            MechanicalObjectCudaVecf_vSum_kernel<<< grid, threads, RED_BSIZE * sizeof(float) >>>(nblocs, (float*)tmp, (const float*)tmp);
            hipMemcpy(res,tmp,sizeof(float),hipMemcpyDeviceToHost);
            */
            hipMemcpy(rtmp,tmp,nblocs*sizeof(float),hipMemcpyDeviceToHost);
            float r = 0.0f;
            for (int i=0; i<nblocs; i++)
                r+=rtmp[i];
            *res = r;
            //myprintf("dot=%f\n",r);
        }
    }
}

int MechanicalObjectCudaVec3f1_vDotTmpSize(unsigned int size)
{
    size *= 4;
    int nblocs = (size+RED_BSIZE-1)/RED_BSIZE;
    if (nblocs > 256) nblocs = 256;
    return nblocs; //(nblocs+3)/4;
}

void MechanicalObjectCudaVec3f1_vDot(unsigned int size, float* res, const void* a, const void* b, void* tmp, float* rtmp)
{
    size *= 4;
    if (size==0)
    {
        *res = 0.0f;
    }
    else
    {
        int nblocs = (size+RED_BSIZE-1)/RED_BSIZE;
        if (nblocs > 256) nblocs = 256;
        dim3 threads(RED_BSIZE,1);
        dim3 grid(nblocs,1);
        //myprintf("size=%d, blocs=%dx%d\n",size,nblocs,RED_BSIZE);
        MechanicalObjectCudaVecf_vDot_kernel<<< grid, threads, RED_BSIZE * sizeof(float) >>>(size, (float*)tmp, (const float*)a, (const float*)b);
        if (nblocs == 1)
        {
            hipMemcpy(res,tmp,sizeof(float),hipMemcpyDeviceToHost);
        }
        else
        {
            /*
            dim3 threads(RED_BSIZE,1);
            dim3 grid(1,1);
            MechanicalObjectCudaVecf_vSum_kernel<<< grid, threads, RED_BSIZE * sizeof(float) >>>(nblocs, (float*)tmp, (const float*)tmp);
            hipMemcpy(res,tmp,sizeof(float),hipMemcpyDeviceToHost);
            */
            hipMemcpy(rtmp,tmp,nblocs*sizeof(float),hipMemcpyDeviceToHost);
            float r = 0.0f;
            for (int i=0; i<nblocs; i++)
                r+=rtmp[i];
            *res = r;
            //myprintf("dot=%f\n",r);
        }
    }
}

#if defined(__cplusplus) && CUDA_VERSION != 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
